#include "hip/hip_runtime.h"
#include "convKernel.h"
#include <stdio.h>


__host__ __device__ int dim(Tensor tensor, int dim) {
    return tensor.dims[dim];
}

__host__ __device__ int stride(Tensor tensor, int dim) {
    return tensor.strides[dim];
}

__host__ __device__ int offset(Tensor tensor, int d0, int d1) {
    return d0 + tensor.strides[0]*d1;
}

__host__ __device__ int offset(Tensor tensor, int d0, int d1, int d2) {
    return d0 + tensor.strides[0]*d1 + tensor.strides[1]*d2;
}

__host__ __device__ int offset(Tensor tensor, int d0, int d1, int d2, int d3) {
    return d0 + tensor.strides[0]*d1 + tensor.strides[1]*d2 + tensor.strides[2]*d3;
}

__host__ __device__ double cellValue(Tensor tensor, int d0, int d1, int d2) {
    return tensor.elements[offset(tensor, d0, d1, d2)];
}

__host__ __device__ double cellValue(Tensor tensor, int d0, int d1, int d2, int d3) {
    return tensor.elements[offset(tensor, d0, d1, d2, d3)];
}

__host__ __device__ void setCellValue(Tensor tensor, double value, int d0, int d1, int d2) {
    tensor.elements[offset(tensor, d0, d1, d2)] = value;
}

__host__ __device__ void setCellValue(Tensor tensor, double value, int d0, int d1, int d2, int d3) {
    tensor.elements[offset(tensor, d0, d1, d2, d3)] = value;
}

// __device__ Tensor cnnSubTensor(const Tensor source, int x, int y, int blockWidth, int blockHeight) {
//     Tensor sub;
//     sub.width = blockWidth;
//     sub.height = blockHeight;
//     sub.depth = source.depth;

//     sub.stride = source.stride;
//     sub.layerStride = source.layerStride;

//     sub.elements = &source.elements[source.stride * blockHeight * y + blockWidth * x];
//     return sub;
// }

__host__ __device__ Tensor tensorSubBlock(const Tensor source, int idx0, int dim0, int idx1, int dim1) {
    Tensor sub;
    sub.dim = 2;
    for (int d=0; d<source.dim; ++d) {
        sub.strides[d] = sub.strides[d];
    }
    sub.dims[0] = dim0;
    sub.dims[1] = dim1;

    sub.elements = &source.elements[offset(source, idx0, idx1)];
    return sub;
};

__host__ __device__ Tensor tensorSubBlock(const Tensor source, int idx0, int dim0, int idx1, int dim1, int idx2, int dim2) {
    Tensor sub;
    sub.dim = 3;
    for (int d=0; d<source.dim; ++d) {
        sub.strides[d] = sub.strides[d];
    }
    sub.dims[0] = dim0;
    sub.dims[1] = dim1;
    sub.dims[2] = dim2;

    sub.elements = &source.elements[offset(source, idx0, idx1, idx2)];
    return sub;
};

__host__ __device__ Tensor tensorSubBlock(const Tensor source,
    int idx0, int dim0,
    int idx1, int dim1,
    int idx2, int dim2,
    int idx3, int dim3) {
    Tensor sub;
    sub.dim = 4;
    for (int d=0; d<source.dim; ++d) {
        sub.strides[d] = sub.strides[d];
    }
    sub.dims[0] = dim0;
    sub.dims[1] = dim1;
    sub.dims[2] = dim2;
    sub.dims[3] = dim3;

    sub.elements = &source.elements[offset(source, idx0, idx1, idx2, idx3)];
    return sub;
};

__host__ __device__ Tensor tensorLayer(const Tensor source, int dim, int idx) {
    if (dim < 1 || dim > source.dim) {
        return Tensor{};
    }

    if (dim == 1) {
        return tensorSubBlock(
            source,
            0, source.dims[0],
            idx, source.dims[1]
        );
    } else if (dim == 2) {
        return tensorSubBlock(
            source,
            0, source.dims[0],
            0, source.dims[1],
            idx, source.dims[2]
        );
    } else if (dim == 4) {
        return tensorSubBlock(
            source,
            0, source.dims[0],
            0, source.dims[1],
            0, source.dims[2],
            idx, source.dims[3]
        );
    }
}

Tensor createDeviceTensor(const Tensor source, bool copy) {
  // Create a new matrix in device memory.
  Tensor tensor;
  tensor.dim = source.dim;
  for (int i=0; i<source.dim; ++i) {
    tensor.dims[i] = source.dims[i];
    tensor.strides[i] = source.strides[i];
  }

  size_t size = tensor.strides[tensor.dim-1] * sizeof(double);
  hipMalloc((void**) &tensor.elements, size);
  if (copy)
    hipMemcpy(tensor.elements, source.elements, size, hipMemcpyHostToDevice);

  return tensor;
}

// Create a matrix in host memory.
Tensor createHostTensor(const TensorDescriptor tensorDescriptor){
  Tensor tensor;
  int stride = 1;

  tensor.dim = tensorDescriptor.dim;
  for (int i=0; i<tensorDescriptor.dim; ++i) {
    tensor.dims[i] = tensorDescriptor.dims[i];
    stride = stride * tensorDescriptor.dims[i];
    tensor.strides[i] = stride;
    printf("Stride %d: %d from %d\n", i, tensor.strides[i], tensorDescriptor.dims[i]);
  }

  size_t size = tensor.strides[tensor.dim-1] * sizeof(double);
  // printf("Creating tensor with dims (%d, %d, %d) and size %zu\n", width, height, depth, size);
  tensor.elements = (double*)malloc(size);

  // printf("Created tensor with dims (%d, %d, %d) and size %zu\n", width, height, depth, size);

  return tensor;
}

__device__ double convolveWithFilter(const Tensor input, const Tensor filter, int x, int y) {
    // using lecture notes as a basis for this function
    double pixelValue = 0.0;

    // note that z is the same for both the filter andand the input
    int width = filter.dims[0];
    int height = filter.dims[1];
    int depth = filter.dims[2];
    int start_x = x - (width/2);
    int start_y = y - (height/2);

    // note that z is the same for both the filter and the input
    for (int z = 0; z < depth; ++z) {
        for(int dy = 0; dy < height; ++dy) {
            for(int dx = 0; dx < width; ++dx) {
                int in_x = start_x + dx;
                int in_y = start_y + dy;
                
                // Verify we are inside the boundaries width and height
                if(in_x > -1 && in_x < input.dims[0]
                    && in_y > -1 && in_y < input.dims[1]) {
                    //NOTE: we flip dy and dx when indexing into the filter in order to get the transpose of it
                    pixelValue += cellValue(input, in_x, in_y, z) * cellValue(filter, dy, dx, z);
                }
            }
        }
    }

    return pixelValue;
}

// __global__ void ConvTiled(const Tensor input, Tensor output, const Tensor filters) {
//     // declare shared
//     __shared__ double filters[64][3][3][3];
//     __shared__ double shared_input[BLOCK_SIZE+1][BLOCK_SIZE+1][3];


//     int threadId = threadIdx.y * blockDim.x + threadIdx.x;
//     int out_x = blockIdx.x * blockDim.x + threadIdx.x;
//     int out_y = blockIdx.y * blockDim.y + threadIdx.y;
//     int filterCount = output.depth;

//     // copy filters and inputs to shared memory
//     for (int out_z = 0; out_z < filterCount; ++out_z) {
//         int k = threadId 
//     }

//     // 

//     // convolve for each filter
//     for (int out_z = 0; out_z < filterCount; ++out_z) {

//         if (out_x < output.width && out_y < output.height) {
//             double pixelValue = convolveWithFilter(input, filters[out_z], out_x, out_y);
//             setCellValue(output, pixelValue, out_x, out_y, out_z);
//         }
//     }

// }


__global__ void Conv(const Tensor input, Tensor output, const Tensor filters) {
    // int threadId = threadIdx.y * blockDim.x + threadIdx.x;
    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;
    int filterCount = output.dims[2];

    for (int out_z = 0; out_z < filterCount; ++out_z) {
        Tensor filter = tensorSubBlock(filters,
            0, filters.dims[0],
            0, filters.dims[1],
            0, filters.dims[2],
            out_z, 1);

        if (out_x < output.dims[0] && out_y < output.dims[1]) {
            double pixelValue = convolveWithFilter(input, filter, out_x, out_y);
            setCellValue(output, pixelValue, out_x, out_y, out_z);
        }
    }
}

__host__ void printTensor(const Tensor source, int x_lim, int y_lim, int z_lim) {
    printf("Tensor Specs:\n");
    printf("Dim: %d\n", source.dim);
    printf("Dims: ");
    for (int i=0; i < source.dim; ++i) {
        printf("%d: %d, ", i, source.dims[i]);
    }
    printf("\nStrides: ");
    for (int i=0; i < source.dim; ++i) {
        printf("%d: %d, ", i, source.strides[i]);
    }
    printf("\n");

    for (int z=0; z < z_lim; ++z) {
        printf("\nDepth=%d", z);
        for (int y=0; y < y_lim; ++y) {
            printf("\n");
            for (int x=0; x < x_lim; ++x) {
                printf("%lf ", cellValue(source, x, y, z));
            }
        }
    }
    printf("\n");
}

__host__ void printTensorDescriptor(const TensorDescriptor source) {
    printf("TensorDescriptor Specs:\n");
    printf("Dim: %d\n", source.dim);
    printf("Dims: ");
    for (int i=0; i < source.dim; ++i) {
        printf("%d, ", source.dims[i]);
    }
    printf("\n");
}