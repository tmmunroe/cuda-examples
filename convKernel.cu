#include "hip/hip_runtime.h"
#include "convKernel.h"
#include <stdio.h>


__host__ __device__ int dim(Tensor tensor, int dim) {
    return tensor.dims[dim];
}

__host__ __device__ int stride(Tensor tensor, int dim) {
    return tensor.strides[dim];
}

__host__ __device__ int offset(Tensor tensor, int d0, int d1) {
    return d0 + tensor.strides[0]*d1;
}

__host__ __device__ int offset(Tensor tensor, int d0, int d1, int d2) {
    return d0 + tensor.strides[0]*d1 + tensor.strides[1]*d2;
}

__host__ __device__ int offset(Tensor tensor, int d0, int d1, int d2, int d3) {
    return d0 + tensor.strides[0]*d1 + tensor.strides[1]*d2 + tensor.strides[2]*d3;
}

__host__ __device__ double cellValue(Tensor tensor, int d0, int d1, int d2) {
    return tensor.elements[offset(tensor, d0, d1, d2)];
}

__host__ __device__ double cellValue(Tensor tensor, int d0, int d1, int d2, int d3) {
    return tensor.elements[offset(tensor, d0, d1, d2, d3)];
}

__host__ __device__ void setCellValue(Tensor tensor, double value, int d0, int d1, int d2) {
    tensor.elements[offset(tensor, d0, d1, d2)] = value;
}

__host__ __device__ void setCellValue(Tensor tensor, double value, int d0, int d1, int d2, int d3) {
    tensor.elements[offset(tensor, d0, d1, d2, d3)] = value;
}

// __device__ Tensor cnnSubTensor(const Tensor source, int x, int y, int blockWidth, int blockHeight) {
//     Tensor sub;
//     sub.width = blockWidth;
//     sub.height = blockHeight;
//     sub.depth = source.depth;

//     sub.stride = source.stride;
//     sub.layerStride = source.layerStride;

//     sub.elements = &source.elements[source.stride * blockHeight * y + blockWidth * x];
//     return sub;
// }

__host__ __device__ Tensor tensorSubBlock(const Tensor source, int idx0, int dim0, int idx1, int dim1) {
    Tensor sub;
    sub.dim = 2;
    for (int d=0; d<source.dim; ++d) {
        sub.strides[d] = source.strides[d];
    }
    sub.dims[0] = dim0;
    sub.dims[1] = dim1;

    sub.elements = &source.elements[offset(source, idx0, idx1)];
    return sub;
};

__host__ __device__ Tensor tensorSubBlock(const Tensor source, int idx0, int dim0, int idx1, int dim1, int idx2, int dim2) {
    Tensor sub;
    sub.dim = 3;
    for (int d=0; d<source.dim; ++d) {
        sub.strides[d] = source.strides[d];
    }
    sub.dims[0] = dim0;
    sub.dims[1] = dim1;
    sub.dims[2] = dim2;

    sub.elements = &source.elements[offset(source, idx0, idx1, idx2)];
    return sub;
};

__host__ __device__ Tensor tensorSubBlock(const Tensor source,
    int idx0, int dim0,
    int idx1, int dim1,
    int idx2, int dim2,
    int idx3, int _dim3) {
    Tensor sub;
    sub.dim = 4;
    for (int d=0; d<source.dim; ++d) {
        sub.strides[d] = source.strides[d];
    }
    sub.dims[0] = dim0;
    sub.dims[1] = dim1;
    sub.dims[2] = dim2;
    sub.dims[3] = _dim3;
	
    //printf("\ntensorSubBlock: (%d, %d) (%d, %d) (%d, %d) (%d, %d), offset: %d\n\n",
		    //idx0, dim0, idx1, dim1, idx2, dim2, idx3, _dim3, offset(source, idx0, idx1, idx2, idx3)
		    //);
    sub.elements = &source.elements[offset(source, idx0, idx1, idx2, idx3)];
    //printf("\nfirst elements: %f, %f, %f\n", sub.elements[0], sub.elements[1], sub.elements[2]);
    return sub;
};

__host__ __device__ Tensor tensorLayer(const Tensor source, int dim, int idx) {
    if (dim < 1 || dim > source.dim) {
        return Tensor{};
    }

    if (dim == 1) {
        return tensorSubBlock(
            source,
            0, source.dims[0],
            idx, 1
        );
    } else if (dim == 2) {
        return tensorSubBlock(
            source,
            0, source.dims[0],
            0, source.dims[1],
            idx, 1
	);
    } else if (dim == 4) {
        return tensorSubBlock(
            source,
            0, source.dims[0],
            0, source.dims[1],
            0, source.dims[2],
            idx, 1
        );
    }
}

Tensor createDeviceTensor(const Tensor source, bool copy) {
  // Create a new matrix in device memory.
  Tensor tensor;
  tensor.dim = source.dim;
  for (int i=0; i<source.dim; ++i) {
    tensor.dims[i] = source.dims[i];
    tensor.strides[i] = source.strides[i];
  }

  size_t size = tensor.strides[tensor.dim-1] * sizeof(double);
  hipMalloc((void**) &tensor.elements, size);
  if (copy)
    hipMemcpy(tensor.elements, source.elements, size, hipMemcpyHostToDevice);

  return tensor;
}

// Create a matrix in host memory.
Tensor createHostTensor(const TensorDescriptor tensorDescriptor){
  Tensor tensor;
  int stride = 1;

  tensor.dim = tensorDescriptor.dim;
  for (int i=0; i<tensorDescriptor.dim; ++i) {
    tensor.dims[i] = tensorDescriptor.dims[i];
    stride = stride * tensorDescriptor.dims[i];
    tensor.strides[i] = stride;
    printf("Stride %d: %d from %d\n", i, tensor.strides[i], tensorDescriptor.dims[i]);
  }

  size_t size = tensor.strides[tensor.dim-1] * sizeof(double);
  // printf("Creating tensor with dims (%d, %d, %d) and size %zu\n", width, height, depth, size);
  tensor.elements = (double*)malloc(size);

  // printf("Created tensor with dims (%d, %d, %d) and size %zu\n", width, height, depth, size);

  return tensor;
}

__device__ double convolveWithFilter(const Tensor input, const Tensor filter, int x, int y) {
    // using lecture notes as a basis for this function
    double pixelValue = 0.0;

    // note that z is the same for both the filter andand the input
    int width = filter.dims[0];
    int height = filter.dims[1];
    int depth = filter.dims[2];
    int input_width = input.dims[0];
    int input_height = input.dims[1];

    int start_x = x - (width/2);
    int start_y = y - (height/2);

    //printf("x: %d, y: %d, start_x: %d, start_y: %d\n", x, y, start_x, start_y);
    // note that z is the same for both the filter and the input
    for (int z = 0; z < depth; ++z) {
        for(int dy = 0; dy < height; ++dy) {
            for(int dx = 0; dx < width; ++dx) {
                int in_x = start_x + dx;
                int in_y = start_y + dy;
                
                // Verify we are inside the boundaries width and height
                if(in_x > -1 && in_x < input_width
                    && in_y > -1 && in_y < input_height) {
                    //NOTE: we flip dy and dx when indexing into the filter in order to get the transpose of it
                    pixelValue += cellValue(input, in_x, in_y, z) * cellValue(filter, dy, dx, z);
                }
            }
        }
    }

//printf("cellvalue: %lf, cellvalue: %lf\n", cellValue(input, x, y, 0), cellValue(filter, 1,1,1));
//printf("returning (%d, %d): %lf... (w: %d, h: %d, d: %d, iw: %d, ih: %d)\n", x, y, pixelValue, width, height, depth, input_width, input_height);
    return pixelValue;
}

// __global__ void ConvTiled(const Tensor input, Tensor output, const Tensor filters) {
//     // declare shared
//     __shared__ double filters[64][3][3][3];
//     __shared__ double shared_input[BLOCK_SIZE+1][BLOCK_SIZE+1][3];


//     int threadId = threadIdx.y * blockDim.x + threadIdx.x;
//     int out_x = blockIdx.x * blockDim.x + threadIdx.x;
//     int out_y = blockIdx.y * blockDim.y + threadIdx.y;
//     int filterCount = output.depth;

//     // copy filters and inputs to shared memory
//     for (int out_z = 0; out_z < filterCount; ++out_z) {
//         int k = threadId 
//     }

//     // 

//     // convolve for each filter
//     for (int out_z = 0; out_z < filterCount; ++out_z) {

//         if (out_x < output.width && out_y < output.height) {
//             double pixelValue = convolveWithFilter(input, filters[out_z], out_x, out_y);
//             setCellValue(output, pixelValue, out_x, out_y, out_z);
//         }
//     }

// }


__global__ void Conv(const Tensor input, Tensor output, const Tensor filters) {
    // int threadId = threadIdx.y * blockDim.x + threadIdx.x;
    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;
    int filterCount = output.dims[2];
    for (int out_z = 0; out_z < filterCount; ++out_z) {
        Tensor filter = tensorLayer(filters, 4, out_z);
	
	if (false && out_x == 0 && out_y == 0 && out_z == 1) {
	printf("Filter %d\n", out_z);
        Tensor filter = tensorLayer(filters, 4, out_z);
	printTensor(filter, 3, 3, 3);
	}
        if (out_x < output.dims[0] && out_y < output.dims[1]) {
            double pixelValue = convolveWithFilter(input, filter, out_x, out_y);
            setCellValue(output, pixelValue, out_x, out_y, out_z);
        }
    }
}

__host__ __device__ void printTensor(const Tensor source, int x_lim, int y_lim, int z_lim) {
    printf("Tensor Specs:\n");
    printf("Dim: %d\n", source.dim);
    printf("Dims: ");
    for (int i=0; i < source.dim; ++i) {
        printf("%d: %d, ", i, source.dims[i]);
    }
    printf("\nStrides: ");
    for (int i=0; i < source.dim; ++i) {
        printf("%d: %d, ", i, source.strides[i]);
    }
    printf("\n");

    for (int z=0; z < z_lim; ++z) {
        printf("\nDepth=%d", z);
        for (int y=0; y < y_lim; ++y) {
            printf("\n");
            for (int x=0; x < x_lim; ++x) {
                printf("%lf ", cellValue(source, x, y, z));
            }
        }
    }
    printf("\n");
}

__host__ void printTensorDescriptor(const TensorDescriptor source) {
    printf("TensorDescriptor Specs:\n");
    printf("Dim: %d\n", source.dim);
    printf("Dims: ");
    for (int i=0; i < source.dim; ++i) {
        printf("%d, ", source.dims[i]);
    }
    printf("\n");
}
