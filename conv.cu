#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <string>
#include <iostream>
#include "timer.h"
#include "convKernel.h"

/*
An input tensor I with dimensions: C, H, W. Each element of I is generated as
follows:
I[c, x, y] = c · (x + y)
Page 4
• A set of convolution filters with dimensions: K, C, FH, FW. Each element of the
filter F is generated as follows:
F[k, c, i, j] = (c + k) · (i + j)
• Dimensions are: H = 1024, W = 1024, C = 3, FW = 3, FH = 3, K = 64.

The output tensor O with dimensions: K,W,H
*/

double seconds2milliseconds(double seconds) {
    return seconds*1000;
}

void fillInput(Tensor tensor) {
    double value;
    for (int c = 0; c < tensor.dims[2]; ++c) {
        for (int y = 0; y < tensor.dims[1]; ++y) {
            for (int x = 0; x < tensor.dims[0]; ++x) {
                value = c*double(x+y);
                setCellValue(tensor, value, x, y, c);
            }
        }
    }
}

void fillFilter(Tensor tensor) {
    double value;
    for (int k = 0; k < tensor.dims[3]; ++k) {
        for (int c = 0; c < tensor.dims[2]; ++c) {
            for (int y = 0; y < tensor.dims[1]; ++y) {
                for (int x = 0; x < tensor.dims[0]; ++x) {
                    value = (c+k)*double(x+y);
                    setCellValue(tensor, value, x, y, c);
                }
            }
        }
    }
}

void fillPaddedInput(Tensor paddedInput, Tensor input, int padding, double padValue) {
    double value;
    for (int c = 0; c < input.dims[2]; ++c) {
        for (int y = 0; y < input.dims[1]; ++y) {
            for (int x = 0; x < input.dims[0]; ++x) {
                value = cellValue(input, x, y, c);
                setCellValue(paddedInput, value, x+padding, y+padding, c);
            }
        }

        for (int y = 0; y < padding; ++y) {
            for (int x = 0; x < padding; ++x) {
                setCellValue(paddedInput, padValue, x, y, c);
                setCellValue(paddedInput, padValue,
                    paddedInput.dims[0] - 1 - x,
                    paddedInput.dims[1] - 1 - y,
                    c
                );
            }
        }
    }
}

void fillOnes(Tensor tensor) {
    for (int c = 0; c < tensor.dims[2]; ++c) {
        for (int y = 0; y < tensor.dims[1]; ++y) {
            for (int x = 0; x < tensor.dims[0]; ++x) {
                setCellValue(tensor, 1.0, x, y, c);
            }
        }
    }
}

void fillFilterOnes(Tensor tensor) {
    for (int k = 0; k < tensor.dims[3]; ++k) {
        for (int c = 0; c < tensor.dims[2]; ++c) {
            for (int y = 0; y < tensor.dims[1]; ++y) {
                for (int x = 0; x < tensor.dims[0]; ++x) {
                    setCellValue(tensor, 1.0, x, y, c, k);
                }
            }
        }
    }
}

double calculateChecksum(Tensor output) {
    double checksum(0);
    for (int z= 0; z < output.dims[2]; ++z) {
        for (int y = 0; y < output.dims[1]; ++y) {
            for (int x = 0; x < output.dims[0]; ++x) {
                checksum += cellValue(output, x, y, z);
            }
        }
    }
    return checksum;
}

void checkTestResults(Tensor output) {
    // check result
    double value;
    double expectedValue;

    int errors = 0;
    for (int c = 0; c < output.dims[2]; ++c) {
        for (int y = 0; y < output.dims[1]; ++y) {
            for (int x = 0; x < output.dims[0]; ++x) {
                if (x > 0 && y > 0 && x < output.dims[0]-1 && y < output.dims[1]-1) {
                    expectedValue = 27;
                } else if (x > 0 && x < output.dims[0]-1) {
                    expectedValue = 18;
                } else if (y > 0 && y < output.dims[1]-1) {
                    expectedValue = 18;
                } else { // both edges are off input grid
                    expectedValue = 12;
                }

                value = cellValue(output, x, y, c);
                if (fabs(value - expectedValue) > 1e-5) {
                    //printf("Error at (%d, %d, %d).. value %lf, expected %lf\n", x, y, c, value, expectedValue);
                    ++errors;
                }
            }
        }
    }

    // report result
    if (errors != 0) {
        std::cout << "Test FAILED with " << errors << " errors" << std::endl;

        printf("Section of output: ");
        printTensor(output, 3, 3, 3);
    } else {
        std::cout << "Test PASSED" << std::endl;
    }
}

int main(int argc, char ** argv) {
    bool isTestCase = false;
    bool verbose = false;
    std::string mode("simple");
    if (argc > 3) {
        isTestCase = std::string("test") == argv[1];
        verbose = std::string("verbose") == argv[2];
        mode = std::string(argv[3]);
    } else if (argc > 2) {
        isTestCase = std::string("test") == argv[1];
        verbose = std::string("verbose") == argv[2];
    } else if (argc > 1) {
        isTestCase = std::string("test") == argv[1];
    }

    double time;

    // tensor specifications
    int padding = 1;
    TensorDescriptor inputDescriptor{.dim=3, .dims={1024, 1024, 3}};
    TensorDescriptor outputDescriptor{.dim=3, .dims={1024, 1024, 64}};
    TensorDescriptor paddedInputDescriptor{.dim=3, 
        .dims={
            inputDescriptor.dims[0]+(padding*2),
            inputDescriptor.dims[1]+(padding*2),
            inputDescriptor.dims[2]
        }};

    const int filterDepth(inputDescriptor.dims[2]);
    const int filterCount(outputDescriptor.dims[2]);
    TensorDescriptor filtersDescriptor{.dim=4, .dims={3, 3, filterDepth, filterCount}};

    if (verbose) {
        printf("\nInput Descriptor: \n");
        printTensorDescriptor(inputDescriptor);
        printf("\nPadded Input Descriptor: \n");
        printTensorDescriptor(paddedInputDescriptor);
        printf("\nOutput Descriptor: \n");
        printTensorDescriptor(outputDescriptor);
        printf("\nFilters Descriptor: \n");
        printTensorDescriptor(filtersDescriptor);

    }

    // create tensors for input, output, and an array of tensors for the filters
    Tensor input = createHostTensor(inputDescriptor);
    Tensor paddedInput = createHostTensor(paddedInputDescriptor);
    Tensor output = createHostTensor(outputDescriptor);
    Tensor filters = createHostTensor(filtersDescriptor);

    printf("Created all tensors\n");
    // initialize input tensor and filters with values
    if (isTestCase) {
        std::cout << "filling with test case values..." << std::endl;
        fillOnes(input);
        fillFilterOnes(filters);
    } else {
        fillInput(input);
        fillFilter(filters);
    }

    fillPaddedInput(paddedInput, input, padding, 0.0);

    if (verbose) {
        printf("\n\nSection of filter: \n");
        printTensor(tensorLayer(filters, 4, 2), 3, 3, 3);

        printf("\n\nSection of input: \n");
        printTensor(input, 3, 3, 3);

        printf("\n\nSection of padded input: \n");
        printTensor(paddedInput, 3, 3, 3);
    }

    // create tensors on device
    Tensor devicePaddedInput = createDeviceTensor(paddedInput, true);
    Tensor deviceOutput = createDeviceTensor(output, false);
    Tensor deviceFilters = createDeviceTensor(filters, true);

    //define dimensions
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(deviceOutput.dims[0]/BLOCK_SIZE, deviceOutput.dims[1]/BLOCK_SIZE);
    hipDeviceSynchronize();

    // Initialize timer  
    initialize_timer();
    start_timer();

    if (mode == "simple") { 
        // simple convolution
        Conv<<<dimGrid, dimBlock>>>(devicePaddedInput, deviceOutput, deviceFilters, padding);
    } else if (mode == "tiled") {
        // tiled convolution
        int filterElementCount = elementsCount(filters);
        
        int inputBlockSize = BLOCK_SIZE+(2*padding);
        int inputElementCount = inputBlockSize*inputBlockSize*paddedInput.dims[2];

        int buffer = 64; // some headroom for allocation

        int sharedMemory = (filterElementCount + inputElementCount + buffer) * sizeof(double);
        ConvTiled<<<dimGrid, dimBlock, sharedMemory>>>(devicePaddedInput, deviceOutput, deviceFilters, padding);
    } else {
        throw std::string("unrecognized mode: " + mode);
    }

    hipDeviceSynchronize();

    // Compute and return elapsed time 
    stop_timer();
    time = elapsed_time();

    // copy to host
    size_t size = output.strides[2] * sizeof(double);
    hipMemcpy(output.elements, deviceOutput.elements, size, hipMemcpyDeviceToHost);

    // check result
    if (isTestCase) {
        printf("Checking test results...\n");
        checkTestResults(output);
    } else {
        double checksum = calculateChecksum(output);
        printf("%0.2lf,%0.3lf\n", checksum, seconds2milliseconds(time));
    }

    // cleanup
    free(input.elements);
    free(paddedInput.elements);
    free(output.elements);
    free(filters.elements);

    hipFree(devicePaddedInput.elements);
    hipFree(deviceOutput.elements);
    hipFree(deviceFilters.elements);
}
